#include "hip/hip_runtime.h"
#include "operator/conv2d.cuh"

namespace lotus {

    /*
    *
    * matrix a, b, and c are row-major
    * 
    * -----------------------------------------------------------------------------------------------------------------------------------------------
    * 
    * tile map:
    * 
    *                                          b_tile                            
    *                                                                               128 floats                    
    *                                         -|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-
    *                                 8 floats |     |     |     |     |     |     |     |     |     |     |     |     |     |     |     |     |
    *                                         -|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-
    * 
    *                           8 floats                     
    *                          -|-----|-     --|-----|-----|-----|-----|-----|-----|-----|-----|-----------------------------------------------|-
    *                           |     |        | t0  |  t1 |  t2 |  t3 | t4  | t5  | t6  | t7  |                                               |
    *                           |-----|       -|-----|-----|-----|-----|-----|-----|-----|-----|                                               |
    *               a_tile      |     |        | t8  |  t9 | t10 | t11 | t12 | t13 | t14 | t15 |                                               |
    *                           |-----|       -|-----|-----|----- warp0 -----|-----|-----|-----|                 warp1                         |
    *                           |     |        | t16 | t17 | t18 | t19 | t20 | t21 | t22 | t23 |                                               |
    *                           |-----|       -|-----|-----|-----|-----|-----|-----|-----|-----|                                               |
    *                           |     |        | t24 | t25 | t26 | t27 | t28 | t29 | t30 | t31 |                                               |
    *                           |-----|       -|-----|-----|-----|-----|-----|-----|-----|-----|-----------------------------------------------|-
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                                               |                                               |
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                 warp2                         |                 warp3                         |
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                                               |                                               |     
    *                           |     |        |                                               |                                               |
    *              128 floats   |-----|       -|-------------------------------------------- block --------------------------------------------|-
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                                               |                                               |
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                 warp4                         |                 warp5                         |
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                                               |                                               |
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|-----------------------------------------------|-----------------------------------------------|-
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                                               |                                               |
    *                           |     |        |                                               |                                               |
    *                           |-----|       -|                                               |                                               |
    *                           |     |        |                 warp6                         |                 warp7                         |
    *                           |-----|       -|                                               |                                               |
    *                           |     |        |                                               |                                               |
    *                          -|-----|-      -|-----------------------------------------------|-----------------------------------------------|-
    * 
    * --------------------------------------------------------------------------------------------------------------------------------------------------
    */



    __global__ void sconv2d(const float* x, 
                            const float* k, 
                            bool use_bias, const float* b, 
                            float* y, 
                            const uint32_t k_num, const uint32_t k_c, const uint32_t k_h, const uint32_t k_w, 
                            const uint32_t x_c, const uint32_t padded_x_h, const uint32_t padded_x_w,    
                            const uint32_t y_c, const uint32_t y_h, const uint32_t y_w,
                            const uint32_t stride_h, const uint32_t stride_w,
                            const uint32_t padding_h, const uint32_t padding_w,
                            ActivationFunction af
                            )
    {
        __shared__ float k_smem[2][128][8];
        __shared__ float x_smem[2][8][128];

        float k_frag[2][8];
        float x_frag[2][8];

        float y_frag[8][8] = {0};


        uint32_t warp_idx = threadIdx.x / 32;

        uint32_t thread_idx_in_warp_w = (threadIdx.x%32) % 8;
        uint32_t thread_idx_in_warp_h = (threadIdx.x%32) / 8;

        uint32_t block_offset_y_w = blockIdx.x*128;
        uint32_t block_offset_y_h = blockIdx.y*128;

        uint32_t thread_offset_blocktile_w = (warp_idx%2)*64 + thread_idx_in_warp_w*8;
        uint32_t thread_offset_blocktile_h = (warp_idx/2)*32 + thread_idx_in_warp_h*8;

        uint32_t thread_offset_y_w = block_offset_y_w + thread_offset_blocktile_w;
        uint32_t thread_offset_y_h = block_offset_y_h + thread_offset_blocktile_h;

        uint32_t thread_offset_sts_k_w = (threadIdx.x%2)*4;
        uint32_t thread_offset_sts_k_h = threadIdx.x/2;

        uint32_t thread_offset_k_h = block_offset_y_h+thread_offset_sts_k_h;
        uint32_t thread_offset_k_w = thread_offset_sts_k_w;

        uint32_t thread_offset_sts_x_w = (threadIdx.x%32)*4;
        uint32_t thread_offset_sts_x_h = threadIdx.x/32;

        uint32_t thread_offset_x_w = block_offset_y_w+thread_offset_sts_x_w;
        uint32_t thread_offset_x_h = thread_offset_sts_x_h;


        uint32_t channel_size = k_h * k_w;
        uint32_t kernel_size = k_c * channel_size;

        uint32_t unpadded_x_w = padded_x_w - 2*padding_w;
        uint32_t unpadded_x_h = padded_x_h - 2*padding_h;

        #pragma unroll
        for(uint32_t i=0; i<4; ++i) {
           
            bool k_guard = thread_offset_k_w+i<kernel_size && thread_offset_k_h<k_num;
            if(k_guard) {
                ldgsts32(&k_smem[0][thread_offset_sts_k_h][thread_offset_sts_k_w+i], k+thread_offset_k_h*kernel_size+thread_offset_k_w+i, 1);
            } else {
                k_smem[0][thread_offset_sts_k_h][thread_offset_sts_k_w+i] = 0.f;
            }
        }


        #pragma unroll
        for(uint32_t i=0; i<4; ++i) {

            uint32_t channel_idx = thread_offset_x_h / channel_size;
            uint32_t row_idx_in_window = (thread_offset_x_h-channel_idx*channel_size) / k_w;
            uint32_t col_idx_in_window = thread_offset_x_h-channel_idx*channel_size-row_idx_in_window*k_w;
            uint32_t row_idx = (thread_offset_x_w+i)/y_w*stride_h+row_idx_in_window;
            uint32_t col_idx = (thread_offset_x_w+i)%y_w*stride_w + col_idx_in_window;

            bool x_guard = thread_offset_x_w+i<y_h*y_w && thread_offset_x_h<kernel_size && (row_idx>=padding_h && row_idx<padded_x_h-padding_h) && (col_idx>=padding_w && col_idx<padded_x_w-padding_w);

            if(x_guard) {
                ldgsts32(&x_smem[0][thread_offset_sts_x_h][thread_offset_sts_x_w+i], x+(row_idx-padding_h)*unpadded_x_w+(col_idx-padding_w)+channel_idx*unpadded_x_h*unpadded_x_w, 1);
            } else {
                x_smem[0][thread_offset_sts_x_h][thread_offset_sts_x_w+i] = 0.f;
            } 
        }

        wait();

        __syncthreads();

        uint32_t smem_load_idx = 0;
        uint32_t smem_store_idx = 1;
       

        for(uint32_t k_step=0; k_step<(kernel_size+7)/8-1; ++k_step) {
            thread_offset_k_w += 8;
            thread_offset_x_h += 8;

            #pragma unroll
            for(uint32_t i=0; i<4; ++i) {
            
                bool k_guard = thread_offset_k_w+i<kernel_size && thread_offset_k_h<k_num;
                if(k_guard) {
                    ldgsts32(&k_smem[smem_store_idx][thread_offset_sts_k_h][thread_offset_sts_k_w+i], k+thread_offset_k_h*kernel_size+thread_offset_k_w+i, 1);
                } else {
                    k_smem[smem_store_idx][thread_offset_sts_k_h][thread_offset_sts_k_w+i] = 0.f;
                }
            }

            #pragma unroll
            for(uint32_t i=0; i<4; ++i) {

                uint32_t channel_idx = thread_offset_x_h / channel_size;
                uint32_t row_idx_in_window = (thread_offset_x_h-channel_idx*channel_size) / k_w;
                uint32_t col_idx_in_window = thread_offset_x_h-channel_idx*channel_size-row_idx_in_window*k_w;
                uint32_t row_idx = (thread_offset_x_w+i)/y_w*stride_h+row_idx_in_window;
                uint32_t col_idx = (thread_offset_x_w+i)%y_w*stride_w + col_idx_in_window;

                bool x_guard = thread_offset_x_w+i<y_h*y_w && thread_offset_x_h<kernel_size && (row_idx>=padding_h && row_idx<padded_x_h-padding_h) && (col_idx>=padding_w && col_idx<padded_x_w-padding_w);

                if(x_guard) {
                    ldgsts32(&x_smem[smem_store_idx][thread_offset_sts_x_h][thread_offset_sts_x_w+i], x+(row_idx-padding_h)*unpadded_x_w+(col_idx-padding_w)+channel_idx*unpadded_x_h*unpadded_x_w, 1);
                } else {
                    x_smem[smem_store_idx][thread_offset_sts_x_h][thread_offset_sts_x_w+i] = 0.f;
                } 
            }


            #pragma unroll
            for(uint32_t i=0; i<8; ++i) {
                k_frag[0][i] = k_smem[smem_load_idx][thread_offset_blocktile_h+i][0];
                x_frag[0][i] = x_smem[smem_load_idx][0][thread_offset_blocktile_w+i];
            }

            uint32_t frag_load_idx = 0;
            uint32_t frag_store_idx = 1;

            #pragma unroll
            for(uint32_t i=0; i<7; ++i) {
                #pragma unroll
                for(uint32_t j=0; j<8; ++j) {
                    k_frag[frag_store_idx][j] = k_smem[smem_load_idx][thread_offset_blocktile_h+j][i+1];
                    x_frag[frag_store_idx][j] = x_smem[smem_load_idx][i+1][thread_offset_blocktile_w+j];
                }
                #pragma unroll
                for(uint32_t h=0; h<8; ++h) {
                    for(uint32_t w=0; w<8; ++w) {
                        y_frag[h][w] += k_frag[frag_load_idx][h]*x_frag[frag_load_idx][w];
                    }
                }
                frag_load_idx ^= 1;
                frag_store_idx ^= 1;
            }
            #pragma unroll
            for(uint32_t h=0; h<8; ++h) {
                for(uint32_t w=0; w<8; ++w) {
                    y_frag[h][w] += k_frag[frag_load_idx][h]*x_frag[frag_load_idx][w];
                }
            }

            wait();
            __syncthreads();

            smem_load_idx ^= 1;
            smem_store_idx ^= 1;
        }

        #pragma unroll
        for(uint32_t i=0; i<8; ++i) {
            k_frag[0][i] = k_smem[smem_load_idx][thread_offset_blocktile_h+i][0];
            x_frag[0][i] = x_smem[smem_load_idx][0][thread_offset_blocktile_w+i];
        }

        uint32_t frag_load_idx = 0;
        uint32_t frag_store_idx = 1;

        #pragma unroll
        for(uint32_t i=0; i<7; ++i) {
            #pragma unroll
            for(uint32_t j=0; j<8; ++j) {
                k_frag[frag_store_idx][j] = k_smem[smem_load_idx][thread_offset_blocktile_h+j][i+1];
                x_frag[frag_store_idx][j] = x_smem[smem_load_idx][i+1][thread_offset_blocktile_w+j];
            }
            #pragma unroll
            for(uint32_t h=0; h<8; ++h) {
                for(uint32_t w=0; w<8; ++w) {
                    y_frag[h][w] += k_frag[frag_load_idx][h]*x_frag[frag_load_idx][w];
                }
            }
            frag_load_idx ^= 1;
            frag_store_idx ^= 1;
        }
        #pragma unroll
        for(uint32_t h=0; h<8; ++h) {
            for(uint32_t w=0; w<8; ++w) {
                y_frag[h][w] += k_frag[frag_load_idx][h]*x_frag[frag_load_idx][w];
            }
        }

        #pragma unroll
        for(uint32_t h=0; h<8; ++h) {
            for(uint32_t w=0; w<8; ++w) {
                uint32_t i = thread_offset_y_h+h;
                uint32_t j = thread_offset_y_w+w;
                if(i<k_num && j<y_h*y_w) {
                    float tmp = y_frag[h][w]+(use_bias?b[i]:0);
                    if(af == ActivationFunction::RELU) {
                        y[i*(y_h*y_w)+j] = tmp>0?tmp:0;
                    } else {
                        y[i*(y_h*y_w)+j] = tmp;
                    }
                    
                }
            }
        }
    }


}