#include "hip/hip_runtime.h"
#include "operator/add.cuh"


namespace lotus {

    __global__ void sadd(const float* x1, const float* x2, float* y, uint32_t size, ActivationFunction af) {
        uint32_t offset = blockIdx.x*256 + threadIdx.x;
        if(offset<size) {
            float tmp = x1[offset] + x2[offset];
            if(af==ActivationFunction::RELU) {
                y[offset] = tmp>0?tmp:0; 
            } else {
                y[offset] = tmp;
            }
        }
    };
                             
}