#include "hip/hip_runtime.h"
#include "operator/fused_gemv_add.cuh"
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>




void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}



bool check(const float *h_x,
           const float *h_a,
           const float *h_b,
           const float *h_y,
           int x_w, int a_h) {
    for (int j = 0; j < a_h; ++j) {
        float sum = 0.f;
        for (int p = 0; p < x_w; ++p) {
            sum += h_x[p] * h_a[j * x_w + p];
        }
        sum += h_b[j];
        if (std::fabs(sum - h_y[j]) / std::fabs(sum) > 1e-5f) {
            printf("y[%d] not match, %f vs %f\n", j, sum, h_y[j]);
            return false;
        }
    }
    

    return true;
}


int main() {

    using namespace lotus;

    int x_w = 5099;
    int a_h = 1024;

    float *h_x, *h_a, *h_y, *h_b;
    hipHostMalloc(&h_x, x_w * sizeof(float));
    hipHostMalloc(&h_a, x_w * a_h * sizeof(float));
    hipHostMalloc(&h_y, a_h * sizeof(float));
    hipHostMalloc(&h_b, a_h * sizeof(float));

    random_init(h_a, x_w * a_h);
    random_init(h_x, x_w);
    random_init(h_b, a_h);

    float *d_x, *d_a, *d_y, *d_b;
    hipMalloc(&d_x, x_w * sizeof(float));
    hipMalloc(&d_a, x_w * a_h * sizeof(float));
    hipMalloc(&d_y, a_h * sizeof(float));
    hipMalloc(&d_b, a_h * sizeof(float));

    hipMemcpy(d_a, h_a, x_w * a_h * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, x_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, a_h * sizeof(float), hipMemcpyHostToDevice);

    StreamPool pool(1);

    sfgemva<<<a_h, 256, 0, pool.Stream()>>>(d_x, d_a, d_b, d_y, x_w, a_h);

    hipMemcpy(h_y, d_y, a_h * sizeof(float), hipMemcpyDeviceToHost);

    bool chk = check(h_x, h_a, h_b, h_y, x_w, a_h);

    printf("vector_y check: %s\n", chk ? "OK" : "Failed");

    hipFree(d_x);
    hipFree(d_a);
    hipFree(d_y);
    hipHostFree(h_x);
    hipHostFree(h_a);
    hipHostFree(h_y);

    return 0;
}

