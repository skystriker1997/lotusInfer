#include "hip/hip_runtime.h"
#include "lotus_utils.hpp"
#include "operator/gemm_tensorcore.cuh"
#include <Eigen/Dense> 




int main() 
{
    using namespace lotus;
    using namespace Eigen;
    using MatrixRowMajor = Matrix <float,Dynamic,Dynamic,RowMajor>;
    int seq = 128;
    int input_features = 128;
    int output_features = 128;

    MatrixRowMajor input = MatrixXf::Random(seq,input_features);
    MatrixRowMajor weight = MatrixXf::Random(output_features,input_features);
    RowVectorXf bias = RowVectorXf::Random(output_features);

    MatrixRowMajor output = (input*weight.transpose()).rowwise() + bias; 

    float *d_a, *d_b, *d_c, *d_bias, *h_c;
    hipMalloc(&d_a, seq * input_features * sizeof(float));
    hipMalloc(&d_b, output_features * input_features * sizeof(float));
    hipMalloc(&d_c, seq * output_features * sizeof(float));
    hipMalloc(&d_bias, output_features * sizeof(float));
    hipHostMalloc(&h_c, seq * output_features * sizeof(float));

    hipMemcpy(d_a, input.data(), seq * input_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, weight.data(), output_features * input_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias.data(), output_features * sizeof(float), hipMemcpyHostToDevice);

    Gemm<<<MakeGemmGrid(seq, output_features), MakeGemmBlock()>>>(d_a, d_b, true, d_bias, d_c, seq, output_features, input_features, ActivationFunction::NONE);
    hipMemcpy(h_c, d_c, seq * output_features * sizeof(float), hipMemcpyDeviceToHost);

    for(uint32_t i=0; i<seq; i++) {
        for(uint32_t j=0; j<output_features; j++) {
            float cuda_result = h_c[i*output_features+j];
            float cpu_result = output(i,j);
            if (std::fabs(cpu_result - cuda_result) / std::fabs(cpu_result) > 0.1) {
                printf("gemm c[%d][%d] not match, %f vs %f\n", i, j, cpu_result, cuda_result);
            }
        }
    }

    return 0;
}