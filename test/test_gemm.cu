#include "hip/hip_runtime.h"
#include "lotus_utils.hpp"
#include "operator/gemm_tensorcore.cuh"
#include <Eigen/Dense> 




int main() 
{
    using namespace lotus;
    using namespace Eigen;
    using MatrixRowMajor = Matrix <float,Dynamic,Dynamic,RowMajor>;
    int seq = 255;
    int input_features = 255;
    int output_features = 255;

    MatrixRowMajor input = MatrixXf(seq,input_features);
    MatrixRowMajor weight = MatrixXf(output_features,input_features);
    RowVectorXf bias = RowVectorXf(output_features);

    for(int i=0; i<seq; ++i) {
        for(int j=0; j<input_features; ++j) {
            if((i*j)%2==0) {
                input(i, j) = -1.f/j+100.f/i;
            } else {
                input(i, j) = 1.f/j-100.f/i;
            }
            
        }
    }

    for(int i=0; i<output_features; ++i) {
        for(int j=0; j<input_features; ++j) {
            if((i*j)%2==0) {
                weight(i, j) = 1.f/j-50.f/i;
            } else {
                weight(i, j) = -1.f/j+50.f/i;
            }            
        }
    }

    for(int i=0; i<output_features; ++i) {
        if(i%2==0) {
            bias(i) = 1;
        } else {
            bias(i) = -1;
        }
    }

    MatrixRowMajor output = (input*weight.transpose()).rowwise() + bias; 

    float *d_a, *d_b, *d_c, *d_bias, *h_c;
    hipMalloc(&d_a, seq * input_features * sizeof(float));
    hipMalloc(&d_b, output_features * input_features * sizeof(float));
    hipMalloc(&d_c, seq * output_features * sizeof(float));
    hipMalloc(&d_bias, output_features * sizeof(float));
    hipHostMalloc(&h_c, seq * output_features * sizeof(float));

    hipMemcpy(d_a, input.data(), seq * input_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, weight.data(), output_features * input_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias.data(), output_features * sizeof(float), hipMemcpyHostToDevice);

    Gemm<<<MakeGemmGrid(seq, output_features), MakeGemmBlock()>>>(d_a, d_b, true, d_bias, d_c, seq, output_features, input_features, ActivationFunction::NONE);
    hipMemcpy(h_c, d_c, seq * output_features * sizeof(float), hipMemcpyDeviceToHost);

    for(uint32_t i=0; i<seq; i++) {
        for(uint32_t j=0; j<output_features; j++) {
            float cuda_result = h_c[i*output_features+j];
            float cpu_result = output(i,j);
            if (std::fabs(cpu_result - cuda_result) / std::fabs(cpu_result) > 1e-5f) {
                printf("Matrix C[%d][%d] not match, %f vs %f\n", i, j, cpu_result, cuda_result);
                return 0;
            }
        }
    }

    printf("Matrix C check OK\n");

    return 0;
}