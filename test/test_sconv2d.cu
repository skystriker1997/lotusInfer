#include "hip/hip_runtime.h"
#include "operator/conv2d.cuh"
#include "xtensor/xarray.hpp"
#include "xtensor/xio.hpp"
#include "xtensor/xview.hpp"
#include "xtensor/xrandom.hpp"



bool check( xt::xarray<float>& x,
            xt::xarray<float>& k,
            xt::xarray<float>& b,
            float* y,
            uint32_t padded_x_h,
            uint32_t padded_x_w,
            uint32_t x_c,
            uint32_t k_num,
            uint32_t k_c,
            uint32_t k_h,
            uint32_t k_w,
            uint32_t stride_h,
            uint32_t stride_w,
            uint32_t y_c,
            uint32_t y_h,
            uint32_t y_w,
            uint32_t padding_h,
            uint32_t padding_w
           ) 
{

    auto sum = [](const float& left, const float& right){return left + right;};
    xt::xarray<float> padded_x = xt::zeros<float>({x_c, padded_x_h, padded_x_w});
    xt::view(padded_x, xt::all(), xt::range(padding_h, padded_x_h-padding_h), xt::range(padding_w, padded_x_w-padding_w)) = x;

    for(uint32_t i=0; i<y_h; i++) {
        for(uint32_t j=0; j<y_w; j++) {
            auto x_frag = xt::view(padded_x, xt::all(), xt::range(i*stride_h, i*stride_h+k_h), xt::range(j*stride_w, j*stride_w+k_w));
            for(uint32_t q=0; q<k_num; ++q) {
                auto k_frag = xt::view(k, xt::range(q, q+1), xt::all(), xt::all(), xt::all());
                auto product = k_frag * x_frag;
                float result = xt::reduce(sum, product, {0,1,2,3})(0) + b(q);
                uint32_t target = q*(y_h*y_w) + i*y_w + j;
                if (std::fabs(result - y[target]) / std::fabs(result) > 1e-5f) {
                    printf("y[%d][%d][%d] not match, %f vs %f\n", q, i, j, result, y[target]);
                    return false;
                }
            }
        }
    }
    return true;
}




int main() 
{
    using namespace lotus;

    uint32_t padded_x_h = 257;
    uint32_t padded_x_w = 257;
    uint32_t x_c = 3;
    uint32_t k_num = 32;
    uint32_t k_c = 3;
    uint32_t k_h = 9;
    uint32_t k_w = 9;
    uint32_t stride_h = 2;
    uint32_t stride_w = 2;;
    uint32_t padding_h = 4;
    uint32_t padding_w = 4;

    uint32_t y_c = k_num;

    uint32_t y_h = (padded_x_h-k_h)/stride_h + 1;
    uint32_t y_w = (padded_x_w-k_w)/stride_w + 1;   

    uint32_t unpadded_x_h = padded_x_h - 2*padding_h;
    uint32_t unpadded_x_w = padded_x_w - 2*padding_w;
   
    xt::random::seed(0);
    xt::xarray<float> x = xt::random::randint<int>({x_c, unpadded_x_h, unpadded_x_w}, 0, 100);
    xt::xarray<float> k = xt::random::randint<int>({k_num, k_c,k_h,k_w}, 0, 100);
    xt::xarray<float> b = xt::random::randint<int>({k_num}, 0, 100);

    float* d_x;
    float* d_k;
    float* d_b;
    float* d_y;
    float* h_y;

    hipMalloc(&d_x, unpadded_x_h*unpadded_x_w*x_c*sizeof(float));
    hipMalloc(&d_k, k_num*k_h*k_w*k_c*sizeof(float));
    hipMalloc(&d_b, k_num*sizeof(float));
    hipMalloc(&d_y, y_c*y_h*y_w*sizeof(float));
    hipHostMalloc(&h_y, y_c*y_h*y_w * sizeof(float));

    hipMemcpy(d_x, x.data(), unpadded_x_h*unpadded_x_w*x_c*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k, k.data(), k_num*k_h*k_w*k_c*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), k_num*sizeof(float), hipMemcpyHostToDevice);

    StreamPool pool(1);

    sconv2d<<<CONV2D_GRID(y_c, y_h, y_w), CONV2D_BLOCK(), 0, pool.Stream()>>>(d_x, 
                                                                              d_k, 
                                                                              true, d_b, 
                                                                              d_y, 
                                                                              k_num, k_c, k_h, k_w, 
                                                                              x_c, padded_x_h, padded_x_w,
                                                                              y_c, y_h, y_w,
                                                                              stride_h, stride_w,
                                                                              padding_h, padding_w,
                                                                              ActivationFunction::NONE
                                                                              );

   

    hipMemcpy(h_y, d_y, y_c*y_h*y_w * sizeof(float), hipMemcpyDeviceToHost);

    bool chk = check(   x,
                        k,
                        b,
                        h_y,
                        padded_x_h,
                        padded_x_w,
                        x_c,
                        k_num,
                        k_c,
                        k_h,
                        k_w,
                        stride_h,
                        stride_w,
                        y_c,
                        y_h,
                        y_w,
                        padding_h,
                        padding_w
                    );

    printf("Cube_Y check: %s\n", chk ? "OK" : "Failed");

    hipFree(d_x);
    hipFree(d_k);
    hipFree(d_b);
    hipFree(d_y);
    hipHostFree(h_y);

    return 0;
}